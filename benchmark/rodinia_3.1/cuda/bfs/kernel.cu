#include "hip/hip_runtime.h"
/*********************************************************************************
Implementing Breadth first search on CUDA using algorithm given in HiPC'07
  paper "Accelerating Large Graph Algorithms on the GPU using CUDA"

Copyright (c) 2008 International Institute of Information Technology - Hyderabad.
All rights reserved.

Permission to use, copy, modify and distribute this software and its documentation for
educational purpose is hereby granted without fee, provided that the above copyright
notice and this permission notice appear in all copies of this software and that you do
not sell the software.

THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR
OTHERWISE.

The CUDA Kernel for Applying BFS on a loaded Graph. Created By Pawan Harish
**********************************************************************************/
#ifndef _KERNEL_H_
#define _KERNEL_H_

__device__ __forceinline__ bool ld_gbl_cg (const bool *addr)
{
    short t;
// #if defined(__LP64__) || defined(_WIN64)
    asm ("ld.global.cg.u8 %0, [%1];" : "=h"(t) : "l"(addr));
// #else
//     asm ("ld.global.cg.u8 %0, [%1];" : "=h"(t) : "r"(addr));
// #endif
    return (bool)t;
}

#endif

__global__ void
Kernel(Node* g_graph_nodes, int* g_graph_edges, bool* g_graph_mask, bool* g_updating_graph_mask, bool *g_graph_visited, int* g_cost, int no_of_nodes)
{
	int tid = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	// bool t = ld_gbl_cg(&g_graph_mask[tid]);
	// asm("ld.global.cg.u8 %0 [%1];" : "=h"(t) : "l"(u): "memory");
	if( tid<no_of_nodes && g_graph_mask[tid])
	// if( tid<no_of_nodes && t)
	{
		g_graph_mask[tid]=false;
		for(int i=g_graph_nodes[tid].starting; i<(g_graph_nodes[tid].no_of_edges + g_graph_nodes[tid].starting); i++)
			{
			int id = g_graph_edges[i];
			// int id = __ldg(&g_graph_edges[i]);
			if(!g_graph_visited[id])
				{
				g_cost[id]=g_cost[tid]+1;
				g_updating_graph_mask[id]=true;
				}
			}
	}
}


