#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

//========================================================================================================================================================================================================200
//	DEFINE/INCLUDE
//========================================================================================================================================================================================================200

//======================================================================================================================================================150
//	COMMON
//======================================================================================================================================================150

#include "../common.h"								// (in main program directory)			needed to recognized input variables

//======================================================================================================================================================150
//	UTILITIES
//======================================================================================================================================================150

#include "../util/cuda/hip/hip_runtime.h"					// (in path specified to compiler)	needed by for device functions
#include "../util/timer/timer.h"					// (in path specified to compiler)	needed by timer

//======================================================================================================================================================150
//	KERNEL
//======================================================================================================================================================150

#include "./kernel_gpu_cuda.cu"						// (in current directory)	GPU kernel, cannot include with header file because of complications with passing of constant memory variables

//======================================================================================================================================================150
//	HEADER
//======================================================================================================================================================150

#include "./kernel_gpu_cuda_wrapper.h"				// (in current directory)

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

//========================================================================================================================================================================================================200
//	KERNEL_GPU_CUDA_WRAPPER FUNCTION
//========================================================================================================================================================================================================200

void
kernel_gpu_cuda_wrapper(record *records,
						long records_mem,
						knode *knodes,
						long knodes_elem,
						long knodes_mem,

						int order,
						long maxheight,
						int count,

						long *currKnode,
						long *offset,
						int *keys,
						record *ans)
{

	//======================================================================================================================================================150
	//	CPU VARIABLES
	//======================================================================================================================================================150

	// timer
	long long time0;
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;

	time0 = get_time();

	//======================================================================================================================================================150
	//	GPU SETUP
	//======================================================================================================================================================150

	//====================================================================================================100
	//	INITIAL DRIVER OVERHEAD
	//====================================================================================================100

	hipDeviceSynchronize();

	//====================================================================================================100
	//	EXECUTION PARAMETERS
	//====================================================================================================100

	int numBlocks;
	numBlocks = count;									// max # of blocks can be 65,535
	int threadsPerBlock;
	threadsPerBlock = order < 1024 ? order : 1024;

	printf("# of blocks = %d, # of threads/block = %d (ensure that device can handle)\n", numBlocks, threadsPerBlock);

	time1 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY				(MALLOC)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	DEVICE IN
	//====================================================================================================100

	//==================================================50
	//	recordsD
	//==================================================50

	record *recordsD;
	hipMalloc((void**)&recordsD, records_mem);
	checkCUDAError("hipMalloc  recordsD");

	//==================================================50
	//	knodesD
	//==================================================50

	knode *knodesD;
	hipMalloc((void**)&knodesD, knodes_mem);
	checkCUDAError("hipMalloc  recordsD");

	//==================================================50
	//	currKnodeD
	//==================================================50

	long *currKnodeD;
	hipMalloc((void**)&currKnodeD, count*sizeof(long));
	checkCUDAError("hipMalloc  currKnodeD");

	//==================================================50
	//	offsetD
	//==================================================50

	long *offsetD;
	hipMalloc((void**)&offsetD, count*sizeof(long));
	checkCUDAError("hipMalloc  offsetD");

	//==================================================50
	//	keysD
	//==================================================50

	int *keysD;
	hipMalloc((void**)&keysD, count*sizeof(int));
	checkCUDAError("hipMalloc  keysD");

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansD
	//==================================================50

	record *ansD;
	hipMalloc((void**)&ansD, count*sizeof(record));
	checkCUDAError("hipMalloc ansD");

	time2 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY
	//======================================================================================================================================================150

	//====================================================================================================100
	//	GPU MEMORY				(MALLOC) COPY IN
	//====================================================================================================100

	//==================================================50
	//	recordsD
	//==================================================50

	hipMemcpy(recordsD, records, records_mem, hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy memD");

	//==================================================50
	//	knodesD
	//==================================================50

	hipMemcpy(knodesD, knodes, knodes_mem, hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy memD");

	//==================================================50
	//	currKnodeD
	//==================================================50

	hipMemcpy(currKnodeD, currKnode, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy currKnodeD");

	//==================================================50
	//	offsetD
	//==================================================50

	hipMemcpy(offsetD, offset, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy offsetD");

	//==================================================50
	//	keysD
	//==================================================50

	hipMemcpy(keysD, keys, count*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy keysD");

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansD
	//==================================================50

	hipMemcpy(ansD, ans, count*sizeof(record), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy ansD");

	time3 = get_time();

	unsigned int err;
	hipModule_t module;
    hipFunction_t kernel;
	dim3 grid( numBlocks, 1, 1);
	dim3 threads( threadsPerBlock, 1, 1);
	void* args[] = {&maxheight,
                    &knodesD,
                    &knodes_elem,
                    &recordsD,
                    &currKnodeD,
                    &offsetD,
					&keysD,
                    &ansD
                    };
	err = hipModuleLoad(&module, "gevo.ptx");
	err = hipModuleGetFunction(&kernel, module, "findK");


	//======================================================================================================================================================150
	// findK kernel
	//======================================================================================================================================================150

	// findK<<<numBlocks, threadsPerBlock>>>(	maxheight,

	// 										knodesD,
	// 										knodes_elem,

	// 										recordsD,

	// 										currKnodeD,
	// 										offsetD,
	// 										keysD,
	// 										ansD);
	hipProfilerStart();
	err = hipModuleLaunchKernel(kernel, grid.x, grid.y, grid.z,
				       threads.x, threads.y, threads.z,
				   	   0, 0, args, 0);


	hipDeviceSynchronize();
	hipProfilerStop();
	checkCUDAError("findK");

	time4 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY (CONTD.)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansD
	//==================================================50

	hipMemcpy(ans, ansD, count*sizeof(record), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy ansD");

	time5 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY DEALLOCATION
	//======================================================================================================================================================150

	hipFree(recordsD);
	hipFree(knodesD);

	hipFree(currKnodeD);
	hipFree(offsetD);
	hipFree(keysD);
	hipFree(ansD);

	time6 = get_time();

	//======================================================================================================================================================150
	//	DISPLAY TIMING
	//======================================================================================================================================================150

	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time6-time0) * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) (time6-time0) / 1000000);

//========================================================================================================================================================================================================200
//	End
//========================================================================================================================================================================================================200

}


void
kernel_gpu_cuda_wrapper_2(	knode *knodes,
							long knodes_elem,
							long knodes_mem,

							int order,
							long maxheight,
							int count,

							long *currKnode,
							long *offset,
							long *lastKnode,
							long *offset_2,
							int *start,
							int *end,
							int *recstart,
							int *reclength)
{

	//======================================================================================================================================================150
	//	CPU VARIABLES
	//======================================================================================================================================================150

	// timer
	long long time0;
	long long time1;
	long long time2;
	long long time3;
	long long time4;
	long long time5;
	long long time6;

	time0 = get_time();

	//======================================================================================================================================================150
	//	GPU SETUP
	//======================================================================================================================================================150

	//====================================================================================================100
	//	INITIAL DRIVER OVERHEAD
	//====================================================================================================100

	hipDeviceSynchronize();

	//====================================================================================================100
	//	EXECUTION PARAMETERS
	//====================================================================================================100

	int numBlocks;
	numBlocks = count;
	int threadsPerBlock;
	threadsPerBlock = order < 1024 ? order : 1024;

	printf("# of blocks = %d, # of threads/block = %d (ensure that device can handle)\n", numBlocks, threadsPerBlock);

	time1 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY				MALLOC
	//======================================================================================================================================================150

	//====================================================================================================100
	//	DEVICE IN
	//====================================================================================================100

	//==================================================50
	//	knodesD
	//==================================================50

	knode *knodesD;
	hipMalloc((void**)&knodesD, knodes_mem);
	checkCUDAError("hipMalloc  recordsD");

	//==================================================50
	//	currKnodeD
	//==================================================50

	long *currKnodeD;
	hipMalloc((void**)&currKnodeD, count*sizeof(long));
	checkCUDAError("hipMalloc  currKnodeD");

	//==================================================50
	//	offsetD
	//==================================================50

	long *offsetD;
	hipMalloc((void**)&offsetD, count*sizeof(long));
	checkCUDAError("hipMalloc  offsetD");

	//==================================================50
	//	lastKnodeD
	//==================================================50

	long *lastKnodeD;
	hipMalloc((void**)&lastKnodeD, count*sizeof(long));
	checkCUDAError("hipMalloc  lastKnodeD");

	//==================================================50
	//	offset_2D
	//==================================================50

	long *offset_2D;
	hipMalloc((void**)&offset_2D, count*sizeof(long));
	checkCUDAError("hipMalloc  offset_2D");

	//==================================================50
	//	startD
	//==================================================50

	int *startD;
	hipMalloc((void**)&startD, count*sizeof(int));
	checkCUDAError("hipMalloc startD");

	//==================================================50
	//	endD
	//==================================================50

	int *endD;
	hipMalloc((void**)&endD, count*sizeof(int));
	checkCUDAError("hipMalloc endD");

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansDStart
	//==================================================50

	int *ansDStart;
	hipMalloc((void**)&ansDStart, count*sizeof(int));
	checkCUDAError("hipMalloc ansDStart");

	//==================================================50
	//	ansDLength
	//==================================================50

	int *ansDLength;
	hipMalloc((void**)&ansDLength, count*sizeof(int));
	checkCUDAError("hipMalloc ansDLength");

	time2 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY
	//======================================================================================================================================================150

	//====================================================================================================100
	//	DEVICE IN
	//====================================================================================================100

	//==================================================50
	//	knodesD
	//==================================================50

	hipMemcpy(knodesD, knodes, knodes_mem, hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy memD");

	//==================================================50
	//	currKnodeD
	//==================================================50

	hipMemcpy(currKnodeD, currKnode, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy currKnodeD");

	//==================================================50
	//	offsetD
	//==================================================50

	hipMemcpy(offsetD, offset, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy offsetD");

	//==================================================50
	//	lastKnodeD
	//==================================================50

	hipMemcpy(lastKnodeD, lastKnode, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy lastKnodeD");

	//==================================================50
	//	offset_2D
	//==================================================50

	hipMemcpy(offset_2D, offset_2, count*sizeof(long), hipMemcpyHostToDevice);
	checkCUDAError("hipMalloc hipMemcpy offset_2D");

	//==================================================50
	//	startD
	//==================================================50

	hipMemcpy(startD, start, count*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy startD");

	//==================================================50
	//	endD
	//==================================================50

	hipMemcpy(endD, end, count*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy endD");

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansDStart
	//==================================================50

	hipMemcpy(ansDStart, recstart, count*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy ansDStart");

	//==================================================50
	//	ansDLength
	//==================================================50

	hipMemcpy(ansDLength, reclength, count*sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy ansDLength");

	time3 = get_time();
	unsigned int err;
	hipModule_t module;
    hipFunction_t kernel;
	dim3 grid( numBlocks, 1, 1);
	dim3 threads( threadsPerBlock, 1, 1);
	void* args[] = {&maxheight,
                    &knodesD,
                    &knodes_elem,
                    &currKnodeD,
                    &offsetD,
                    &lastKnodeD,
					&offset_2D,
					&startD,
					&endD,
					&ansDStart,
					&ansDLength
                    };
	err = hipModuleLoad(&module, "gevo.ptx");
	err = hipModuleGetFunction(&kernel, module, "findRangeK");

	//======================================================================================================================================================150
	//	KERNEL
	//======================================================================================================================================================150

	// [GPU] findRangeK kernel
	// findRangeK<<<numBlocks, threadsPerBlock>>>(	maxheight,
	// 											knodesD,
	// 											knodes_elem,

	// 											currKnodeD,
	// 											offsetD,
	// 											lastKnodeD,
	// 											offset_2D,
	// 											startD,
	// 											endD,
	// 											ansDStart,
	// 											ansDLength);
	hipProfilerStart();
	err = hipModuleLaunchKernel(kernel, grid.x, grid.y, grid.z,
				       threads.x, threads.y, threads.z,
				   	   0, 0, args, 0);

	hipDeviceSynchronize();
	hipProfilerStop();
	checkCUDAError("findRangeK");

	time4 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY			COPY (CONTD.)
	//======================================================================================================================================================150

	//====================================================================================================100
	//	DEVICE IN/OUT
	//====================================================================================================100

	//==================================================50
	//	ansDStart
	//==================================================50

	hipMemcpy(recstart, ansDStart, count*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy ansDStart");

	//==================================================50
	//	ansDLength
	//==================================================50

	hipMemcpy(reclength, ansDLength, count*sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy ansDLength");

	time5 = get_time();

	//======================================================================================================================================================150
	//	GPU MEMORY DEALLOCATION
	//======================================================================================================================================================150

	hipFree(knodesD);

	hipFree(currKnodeD);
	hipFree(offsetD);
	hipFree(lastKnodeD);
	hipFree(offset_2D);
	hipFree(startD);
	hipFree(endD);
	hipFree(ansDStart);
	hipFree(ansDLength);

	time6 = get_time();

	//======================================================================================================================================================150
	//	DISPLAY TIMING
	//======================================================================================================================================================150

	printf("Time spent in different stages of GPU_CUDA KERNEL:\n");

	printf("%15.12f s, %15.12f % : GPU: SET DEVICE / DRIVER INIT\n",	(float) (time1-time0) / 1000000, (float) (time1-time0) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: ALO\n", 					(float) (time2-time1) / 1000000, (float) (time2-time1) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: COPY IN\n",					(float) (time3-time2) / 1000000, (float) (time3-time2) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU: KERNEL\n",						(float) (time4-time3) / 1000000, (float) (time4-time3) / (float) (time6-time0) * 100);

	printf("%15.12f s, %15.12f % : GPU MEM: COPY OUT\n",				(float) (time5-time4) / 1000000, (float) (time5-time4) / (float) (time6-time0) * 100);
	printf("%15.12f s, %15.12f % : GPU MEM: FRE\n", 					(float) (time6-time5) / 1000000, (float) (time6-time5) / (float) (time6-time0) * 100);

	printf("Total time:\n");
	printf("%.12f s\n", 												(float) (time6-time0) / 1000000);

}

//========================================================================================================================================================================================================200
//	END
//========================================================================================================================================================================================================200

#ifdef __cplusplus
}
#endif
