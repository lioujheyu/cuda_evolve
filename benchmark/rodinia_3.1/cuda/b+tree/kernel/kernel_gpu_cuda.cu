#include "hip/hip_runtime.h"
//========================================================================================================================================================================================================200
//	findK function
//========================================================================================================================================================================================================200

__global__ void
findK(	long height,
		knode *knodesD,
		long knodes_elem,
		record *recordsD,

		long *currKnodeD,
		long *offsetD,
		int *keysD,
		record *ansD)
{

	// private thread IDs
	int thid = threadIdx.x;
	int bid = blockIdx.x;

	// // processtree levels
	// int i;
	// for(i = 0; i < height; i++){

	// 	// if value is between the two keys
	// 	if((knodesD[currKnodeD[bid]].keys[thid]) <= keysD[bid] && (knodesD[currKnodeD[bid]].keys[thid+1] > keysD[bid])){
	// 		// this conditional statement is inserted to avoid crush due to but in original code
	// 		// "offset[bid]" calculated below that addresses knodes[] in the next iteration goes outside of its bounds cause segmentation fault
	// 		// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
	// 		if(knodesD[offsetD[bid]].indices[thid] < knodes_elem){
	// 			offsetD[bid] = knodesD[offsetD[bid]].indices[thid];
	// 		}
	// 	}
	// 	__syncthreads();

	// 	// set for next tree level
	// 	if(thid==0){
	// 		currKnodeD[bid] = offsetD[bid];
	// 	}
	// 	__syncthreads();

	// }

	// //At this point, we have a candidate leaf node which may contain
	// //the target record.  Check each key to hopefully find the record
	// if(knodesD[currKnodeD[bid]].keys[thid] == keysD[bid]){
	// 	ansD[bid].value = recordsD[knodesD[currKnodeD[bid]].indices[thid]].value;
	// }

	ansD[bid].value = keysD[bid];

}

//========================================================================================================================================================================================================200
//	findRangeK function
//========================================================================================================================================================================================================200
__global__ void
findRangeK(	long height,

			knode *knodesD,
			long knodes_elem,

			long *currKnodeD,
			long *offsetD,
			long *lastKnodeD,
			long *offset_2D,
			int *startD,
			int *endD,
			int *RecstartD,
			int *ReclenD)
{

	// private thread IDs
	int thid = threadIdx.x;
	int bid = blockIdx.x;

	// ???
	int i;
	for(i = 0; i < height; i++){

		if((knodesD[currKnodeD[bid]].keys[thid] <= startD[bid]) && (knodesD[currKnodeD[bid]].keys[thid+1] > startD[bid])){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[currKnodeD[bid]].indices[thid] < knodes_elem){
				offsetD[bid] = knodesD[currKnodeD[bid]].indices[thid];
			}
		}
		if((knodesD[lastKnodeD[bid]].keys[thid] <= endD[bid]) && (knodesD[lastKnodeD[bid]].keys[thid+1] > endD[bid])){
			// this conditional statement is inserted to avoid crush due to but in original code
			// "offset_2[bid]" calculated below that later addresses part of knodes goes outside of its bounds cause segmentation fault
			// more specifically, values saved into knodes->indices in the main function are out of bounds of knodes that they address
			if(knodesD[lastKnodeD[bid]].indices[thid] < knodes_elem){
				offset_2D[bid] = knodesD[lastKnodeD[bid]].indices[thid];
			}
		}
		__syncthreads();

		// set for next tree level
		if(thid==0){
			currKnodeD[bid] = offsetD[bid];
			lastKnodeD[bid] = offset_2D[bid];
		}
		__syncthreads();
	}

	// Find the index of the starting record
	if(knodesD[currKnodeD[bid]].keys[thid] == startD[bid]){
		RecstartD[bid] = knodesD[currKnodeD[bid]].indices[thid];
	}
	__syncthreads();

	// Find the index of the ending record
	if(knodesD[lastKnodeD[bid]].keys[thid] == endD[bid]){
		ReclenD[bid] = knodesD[lastKnodeD[bid]].indices[thid] - RecstartD[bid]+1;
	}

}

//========================================================================================================================================================================================================200
//	End
//========================================================================================================================================================================================================200
