#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  lud.cu
 *
 *    Description:  The main wrapper for the suite
 *
 *        Version:  1.0
 *        Created:  10/22/2009 08:40:34 PM
 *       Revision:  none
 *       Compiler:  gcc
 *
 *         Author:  Liang Wang (lw2aw), lw2aw@virginia.edu
 *        Company:  CS@UVa
 *
 * =====================================================================================
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <getopt.h>
#include <stdlib.h>
#include <assert.h>
#include <fstream>
#include <sstream>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>

#include "common.h"

#ifdef RD_WG_SIZE_0_0
        #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
        #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
        #define BLOCK_SIZE RD_WG_SIZE
#else
        #define BLOCK_SIZE 16
#endif

static int do_verify = 0;

static struct option long_options[] = {
  /* name, has_arg, flag, val */
  {"input", 1, NULL, 'i'},
  {"size", 1, NULL, 's'},
  {"verify", 0, NULL, 'v'},
  {0,0,0,0}
};

__global__ void
lud_diagonal(float *m, int matrix_dim, int offset)
{
  int i,j;
  __shared__ float shadow[BLOCK_SIZE][BLOCK_SIZE];

  int array_offset = offset*matrix_dim+offset;
  for(i=0; i < BLOCK_SIZE; i++){
    shadow[i][threadIdx.x]=m[array_offset+threadIdx.x];
    array_offset += matrix_dim;
  }
  __syncthreads();
  for(i=0; i < BLOCK_SIZE-1; i++) {

    if (threadIdx.x>i){
      for(j=0; j < i; j++)
        shadow[threadIdx.x][i] -= shadow[threadIdx.x][j]*shadow[j][i];
      shadow[threadIdx.x][i] /= shadow[i][i];
    }

    __syncthreads();
    if (threadIdx.x>i){

      for(j=0; j < i+1; j++)
        shadow[i+1][threadIdx.x] -= shadow[i+1][j]*shadow[j][threadIdx.x];
    }
    __syncthreads();
  }

  /*
     The first row is not modified, it
     is no need to write it back to the
     global memory

   */
  array_offset = (offset+1)*matrix_dim+offset;
  for(i=1; i < BLOCK_SIZE; i++){
    m[array_offset+threadIdx.x]=shadow[i][threadIdx.x];
    array_offset += matrix_dim;
  }
}

__global__ void
lud_perimeter(float *m, int matrix_dim, int offset)
{
  __shared__ float dia[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i,j, array_offset;
  int idx;

  if (threadIdx.x < BLOCK_SIZE) {
    idx = threadIdx.x;

    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE/2; i++){
      dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }

    array_offset = offset*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_row[i][idx]=m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx];
      array_offset += matrix_dim;
    }

  } else {
    idx = threadIdx.x-BLOCK_SIZE;

    array_offset = (offset+BLOCK_SIZE/2)*matrix_dim+offset;
    for (i=BLOCK_SIZE/2; i < BLOCK_SIZE; i++){
      dia[i][idx]=m[array_offset+idx];
      array_offset += matrix_dim;
    }

    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for (i=0; i < BLOCK_SIZE; i++) {
      peri_col[i][idx] = m[array_offset+idx];
      array_offset += matrix_dim;
    }

  }
  __syncthreads();

/* this version works ok on hardware, but not gpgpusim
 **************************************************************
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
    }


    array_offset = (offset+1)*matrix_dim+offset;
    for(i=1; i < BLOCK_SIZE; i++){
      m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
      array_offset += matrix_dim;
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
      peri_col[idx][i] /= dia[i][i];
    }

    __syncthreads();

    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for(i=0; i < BLOCK_SIZE; i++){
      m[array_offset+idx] =  peri_col[i][idx];
      array_offset += matrix_dim;
    }
  }
***************************************************************/
  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    for(i=1; i < BLOCK_SIZE; i++){
      for (j=0; j < i; j++)
        peri_row[i][idx]-=dia[i][j]*peri_row[j][idx];
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    for(i=0; i < BLOCK_SIZE; i++){
      for(j=0; j < i; j++)
        peri_col[idx][i]-=peri_col[idx][j]*dia[j][i];
      peri_col[idx][i] /= dia[i][i];
    }
  }

  __syncthreads();

  if (threadIdx.x < BLOCK_SIZE) { //peri-row
    idx=threadIdx.x;
    array_offset = (offset+1)*matrix_dim+offset;
    for(i=1; i < BLOCK_SIZE; i++){
      m[array_offset+(blockIdx.x+1)*BLOCK_SIZE+idx] = peri_row[i][idx];
      array_offset += matrix_dim;
    }
  } else { //peri-col
    idx=threadIdx.x - BLOCK_SIZE;
    array_offset = (offset+(blockIdx.x+1)*BLOCK_SIZE)*matrix_dim+offset;
    for(i=0; i < BLOCK_SIZE; i++){
      m[array_offset+idx] =  peri_col[i][idx];
      array_offset += matrix_dim;
    }
  }

}

__global__ void
lud_internal(float *m, int matrix_dim, int offset)
{
  __shared__ float peri_row[BLOCK_SIZE][BLOCK_SIZE];
  __shared__ float peri_col[BLOCK_SIZE][BLOCK_SIZE];

  int i;
  float sum;

  int global_row_id = offset + (blockIdx.y+1)*BLOCK_SIZE;
  int global_col_id = offset + (blockIdx.x+1)*BLOCK_SIZE;

  peri_row[threadIdx.y][threadIdx.x] = m[(offset+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x];
  peri_col[threadIdx.y][threadIdx.x] = m[(global_row_id+threadIdx.y)*matrix_dim+offset+threadIdx.x];

  __syncthreads();

  sum = 0;
  for (i=0; i < BLOCK_SIZE; i++)
    sum += peri_col[threadIdx.y][i] * peri_row[i][threadIdx.x];
  m[(global_row_id+threadIdx.y)*matrix_dim+global_col_id+threadIdx.x] -= sum;


}


void lud_cuda(float *m, int matrix_dim)
{
  int i=0;
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  float *m_debug = (float*)malloc(matrix_dim*matrix_dim*sizeof(float));

  unsigned int err;
  err = hipInit(0);
	hipModule_t module;
  hipFunction_t lud_diagonal_kernel, lud_perimeter_kernel, lud_internal_kernel;
  err = hipModuleLoad(&module, "a.ptx");
	err = hipModuleGetFunction(&lud_diagonal_kernel, module, "_Z12lud_diagonalPfii");
  err = hipModuleGetFunction(&lud_perimeter_kernel, module, "_Z13lud_perimeterPfii");
  err = hipModuleGetFunction(&lud_internal_kernel, module, "_Z12lud_internalPfii");
  void* args[] = { &m, &matrix_dim, &i };

  hipProfilerStart();
  for (i=0; i < matrix_dim-BLOCK_SIZE; i += BLOCK_SIZE) {
      // lud_diagonal<<<1, BLOCK_SIZE>>>(m, matrix_dim, i);
      err = hipModuleLaunchKernel(
        lud_diagonal_kernel,
        1, 1, 1,
        BLOCK_SIZE, 1, 1,
        0, 0, args, 0
      );
      // lud_perimeter<<<(matrix_dim-i)/BLOCK_SIZE-1, BLOCK_SIZE*2>>>(m, matrix_dim, i);
      err = hipModuleLaunchKernel(
        lud_perimeter_kernel,
        (matrix_dim-i)/BLOCK_SIZE-1, 1, 1,
        BLOCK_SIZE*2, 1, 1,
        0, 0, args, 0
      );
      dim3 dimGrid((matrix_dim-i)/BLOCK_SIZE-1, (matrix_dim-i)/BLOCK_SIZE-1);
      // lud_internal<<<dimGrid, dimBlock>>>(m, matrix_dim, i);
      err = hipModuleLaunchKernel(
        lud_internal_kernel,
        dimGrid.x, dimGrid.y, 1,
        dimBlock.x, dimBlock.y, 1,
        0, 0, args, 0
      );
  }
  // lud_diagonal<<<1,BLOCK_SIZE>>>(m, matrix_dim, i);
  err = hipModuleLaunchKernel(
        lud_diagonal_kernel,
        1, 1, 1,
        BLOCK_SIZE, 1, 1,
        0, 0, args, 0
      );
  hipDeviceSynchronize();
  hipProfilerStop();
}



int
main ( int argc, char *argv[] )
{
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);

  int matrix_dim = 32; /* default matrix_dim */
  int opt, option_index=0;
  func_ret_t ret;
  const char *input_file = NULL;
  float *m, *d_m, *mm;
  stopwatch sw;

  while ((opt = getopt_long(argc, argv, "::vs:i:",
                            long_options, &option_index)) != -1 ) {
    switch(opt){
    case 'i':
      input_file = optarg;
      break;
    case 'v':
      do_verify = 1;
      break;
    case 's':
      matrix_dim = atoi(optarg);
      printf("Generate input matrix internally, size =%d\n", matrix_dim);
      // fprintf(stderr, "Currently not supported, use -i instead\n");
      // fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
      // exit(EXIT_FAILURE);
      break;
    case '?':
      fprintf(stderr, "invalid option\n");
      break;
    case ':':
      fprintf(stderr, "missing argument\n");
      break;
    default:
      fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n",
	      argv[0]);
      exit(EXIT_FAILURE);
    }
  }

  if ( (optind < argc) || (optind == 1)) {
    fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  if (input_file) {
    printf("Reading matrix from file %s\n", input_file);
    ret = create_matrix_from_file(&m, input_file, &matrix_dim);
    if (ret != RET_SUCCESS) {
      m = NULL;
      fprintf(stderr, "error create matrix from file %s\n", input_file);
      exit(EXIT_FAILURE);
    }
  }
  else if (matrix_dim) {
    printf("Creating matrix internally size=%d\n", matrix_dim);
    ret = create_matrix(&m, matrix_dim);
    if (ret != RET_SUCCESS) {
      m = NULL;
      fprintf(stderr, "error create matrix internally size=%d\n", matrix_dim);
      exit(EXIT_FAILURE);
    }
  }


  else {
    printf("No input file specified!\n");
    exit(EXIT_FAILURE);
  }

  if (do_verify){
    printf("Before LUD\n");
    // print_matrix(m, matrix_dim);
    matrix_duplicate(m, &mm, matrix_dim);
  }

  hipMalloc((void**)&d_m,
             matrix_dim*matrix_dim*sizeof(float));

  /* beginning of timing point */
  stopwatch_start(&sw);
  hipMemcpy(d_m, m, matrix_dim*matrix_dim*sizeof(float),
	     hipMemcpyHostToDevice);

  lud_cuda(d_m, matrix_dim);

  hipMemcpy(m, d_m, matrix_dim*matrix_dim*sizeof(float),
	     hipMemcpyDeviceToHost);

  /* end of timing point */
  stopwatch_stop(&sw);
  printf("Time consumed(ms): %lf\n", 1000*get_interval_by_sec(&sw));

  hipFree(d_m);

  printf("Dumping the result...\n");
    std::ofstream outf;
    outf.open("output");
    std::stringstream tempstr;

    for (int i=0; i<matrix_dim; i++)
      for (int j=0; j<matrix_dim; j++)
        tempstr << m[i*matrix_dim+j] << "\n";

    outf.write(tempstr.str().c_str(), tempstr.str().length());
    outf.close();

  if (do_verify){
    printf("After LUD\n");
    // print_matrix(m, matrix_dim);
    printf(">>>Verify<<<<\n");
    lud_verify(mm, m, matrix_dim);
    free(mm);
  }

  free(m);

  return EXIT_SUCCESS;
}				/* ----------  end of function main  ---------- */